
#include <hip/hip_runtime.h>
#if defined(HYPRE_USE_GPU)
#include <stdio.h>
#include <cuda_runtime.h>
#include <cublas_v2.h>
#include "_hypre_utilities.h"
#define BlockSize 64
#define MaxSpace 100
extern "C"{
__global__
void VecScaleKernelText(HYPRE_Complex *__restrict__ u, const HYPRE_Complex *__restrict__ v, const HYPRE_Complex *__restrict__ l1_norm, hypre_int num_rows){
  hypre_int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i<num_rows){
    u[i]+=__ldg(v+i)/__ldg(l1_norm+i);
  }
}
}

extern "C"{
__global__
void VecScaleKernel(HYPRE_Complex *__restrict__ u, const HYPRE_Complex *__restrict__ v, const HYPRE_Complex * __restrict__ l1_norm, hypre_int num_rows){
  hypre_int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i<num_rows){
    u[i]+=v[i]/l1_norm[i];
  }
}
}

extern "C"{
void VecScale(HYPRE_Complex *u, HYPRE_Complex *v, HYPRE_Complex *l1_norm, hypre_int num_rows,cudaStream_t s){
  PUSH_RANGE_PAYLOAD("VECSCALE",1,num_rows);
  const hypre_int tpb=64;
  hypre_int num_blocks=num_rows/tpb+1;
#ifdef CATCH_LAUNCH_ERRORS
  hypre_CheckErrorDevice(cudaPeekAtLastError());
  hypre_CheckErrorDevice(cudaDeviceSynchronize());
#endif
  MemPrefetchSized(l1_norm,num_rows*sizeof(HYPRE_Complex),HYPRE_DEVICE,s);
  VecScaleKernel<<<num_blocks,tpb,0,s>>>(u,v,l1_norm,num_rows);
#ifdef CATCH_LAUNCH_ERRORS    
  hypre_CheckErrorDevice(cudaPeekAtLastError());
  hypre_CheckErrorDevice(cudaDeviceSynchronize());
#endif
  hypre_CheckErrorDevice(cudaStreamSynchronize(s));
  POP_RANGE;
}
}


extern "C"{

__global__
void VecCopyKernel(HYPRE_Complex* __restrict__ tgt, const HYPRE_Complex* __restrict__ src, hypre_int size){
  hypre_int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i<size) tgt[i]=src[i];
}
void VecCopy(HYPRE_Complex* tgt, const HYPRE_Complex* src, hypre_int size,cudaStream_t s){
  hypre_int tpb=64;
  hypre_int num_blocks=size/tpb+1;
  PUSH_RANGE_PAYLOAD("VecCopy",5,size);
  //MemPrefetch(tgt,0,s);
  //MemPrefetch(src,0,s);
  VecCopyKernel<<<num_blocks,tpb,0,s>>>(tgt,src,size);
  //hypre_CheckErrorDevice(cudaStreamSynchronize(s));
  POP_RANGE;
}
}
extern "C"{

__global__
void VecSetKernel(HYPRE_Complex* __restrict__ tgt, const HYPRE_Complex value,hypre_int size){
  hypre_int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i<size) tgt[i]=value;
}
void VecSet(HYPRE_Complex* tgt, hypre_int size, HYPRE_Complex value, cudaStream_t s){
  hypre_int tpb=64;
  //cudaDeviceSynchronize();
  MemPrefetchSized(tgt,size*sizeof(HYPRE_Complex),HYPRE_DEVICE,s);
  hypre_int num_blocks=size/tpb+1;
  VecSetKernel<<<num_blocks,tpb,0,s>>>(tgt,value,size);
  cudaStreamSynchronize(s);
  //cudaDeviceSynchronize();
}
}
extern "C"{
__global__
void  PackOnDeviceKernel(HYPRE_Complex* __restrict__ send_data,const HYPRE_Complex* __restrict__ x_local_data, const hypre_int* __restrict__ send_map, hypre_int begin,hypre_int end){
  hypre_int i = begin+blockIdx.x * blockDim.x + threadIdx.x;
  if (i<end){
    send_data[i-begin]=x_local_data[send_map[i]];
  }
}
void PackOnDevice(HYPRE_Complex *send_data,HYPRE_Complex *x_local_data, hypre_int *send_map, hypre_int begin,hypre_int end,cudaStream_t s){
  if ((end-begin)<=0) return;
  hypre_int tpb=64;
  hypre_int num_blocks=(end-begin)/tpb+1;
#ifdef CATCH_LAUNCH_ERRORS
  hypre_CheckErrorDevice(cudaPeekAtLastError());
  hypre_CheckErrorDevice(cudaDeviceSynchronize());
#endif
  PackOnDeviceKernel<<<num_blocks,tpb,0,s>>>(send_data,x_local_data,send_map,begin,end);
#ifdef CATCH_LAUNCH_ERRORS
  hypre_CheckErrorDevice(cudaPeekAtLastError());
  hypre_CheckErrorDevice(cudaDeviceSynchronize());
#endif
  PUSH_RANGE("PACK_PREFETCH",1);
#ifndef HYPRE_GPU_USE_PINNED
  MemPrefetchSized((void*)send_data,(end-begin)*sizeof(HYPRE_Complex),cudaCpuDeviceId,s);
#endif
  POP_RANGE;
  //hypre_CheckErrorDevice(cudaStreamSynchronize(s));
}
}

// Scale vector by scalar

extern "C"{
__global__
void VecScaleScalarKernel(HYPRE_Complex *__restrict__ u, const HYPRE_Complex alpha ,hypre_int num_rows){
  hypre_int i = blockIdx.x * blockDim.x + threadIdx.x;
  //if (i<5) printf("DEVICE %d %lf %lf %lf\n",i,u[i],v[i],l1_norm[i]);
  if (i<num_rows){
    u[i]*=alpha;
    //if (i==0) printf("Diff Device %d %lf %lf %lf\n",i,u[i],v[i],l1_norm[i]);
  }
}
}
extern "C"{
hypre_int VecScaleScalar(HYPRE_Complex *u, const HYPRE_Complex alpha,  hypre_int num_rows,cudaStream_t s){
  PUSH_RANGE("SEQVECSCALE",4);
  hypre_int num_blocks=num_rows/64+1;

#ifdef CATCH_LAUNCH_ERRORS
  hypre_CheckErrorDevice(cudaPeekAtLastError());
  hypre_CheckErrorDevice(cudaDeviceSynchronize());
#endif
  VecScaleScalarKernel<<<num_blocks,64,0,s>>>(u,alpha,num_rows);
#ifdef CATCH_LAUNCH_ERRORS
  hypre_CheckErrorDevice(cudaPeekAtLastError());
  hypre_CheckErrorDevice(cudaDeviceSynchronize());
#endif
  hypre_CheckErrorDevice(cudaStreamSynchronize(s));
  POP_RANGE;
  return 0;
}
}


extern "C"{
  __global__
void SpMVCudaKernel(HYPRE_Complex* __restrict__ y,HYPRE_Complex alpha, const HYPRE_Complex* __restrict__ A_data, const hypre_int* __restrict__ A_i, const hypre_int* __restrict__ A_j, const HYPRE_Complex* __restrict__ x, HYPRE_Complex beta, hypre_int num_rows)
{
  hypre_int i= blockIdx.x * blockDim.x + threadIdx.x;
  if (i<num_rows){
    HYPRE_Complex temp = 0.0;
    hypre_int jj;
    for (jj = A_i[i]; jj < A_i[i+1]; jj++){
      hypre_int ajj=A_j[jj];
      temp += A_data[jj] * x[ajj];
    }
    y[i] =y[i]*beta+alpha*temp;
  }
}

  __global__
void SpMVCudaKernelZB(HYPRE_Complex* __restrict__ y,HYPRE_Complex alpha, const HYPRE_Complex* __restrict__ A_data, const hypre_int* __restrict__ A_i, const hypre_int* __restrict__ A_j, const HYPRE_Complex* __restrict__ x, hypre_int num_rows)
{
  hypre_int i= blockIdx.x * blockDim.x + threadIdx.x;
  if (i<num_rows){
    HYPRE_Complex temp = 0.0;
    hypre_int jj;
    for (jj = A_i[i]; jj < A_i[i+1]; jj++){
      hypre_int ajj=A_j[jj];
      temp += A_data[jj] * x[ajj];
    }
    y[i] = alpha*temp;
  }
}
void SpMVCuda(hypre_int num_rows,HYPRE_Complex alpha, HYPRE_Complex *A_data,hypre_int *A_i, hypre_int *A_j, HYPRE_Complex *x, HYPRE_Complex beta, HYPRE_Complex *y){
  hypre_int num_threads=64;
  hypre_int num_blocks=num_rows/num_threads+1;
#ifdef CATCH_LAUNCH_ERRORS
  hypre_CheckErrorDevice(cudaPeekAtLastError());
  hypre_CheckErrorDevice(cudaDeviceSynchronize());
#endif    
  if (beta==0.0)
    SpMVCudaKernelZB<<<num_blocks,num_threads>>>(y,alpha,A_data,A_i,A_j,x,num_rows);
  else
    SpMVCudaKernel<<<num_blocks,num_threads>>>(y,alpha,A_data,A_i,A_j,x,beta,num_rows);
#ifdef CATCH_LAUNCH_ERRORS
  hypre_CheckErrorDevice(cudaPeekAtLastError());
  hypre_CheckErrorDevice(cudaDeviceSynchronize());
#endif

}
}
extern "C"{
__global__
void CompileFlagSafetyCheck(hypre_int actual){
#ifdef __CUDA_ARCH__
  hypre_int cudarch=__CUDA_ARCH__;
  if (cudarch!=actual){
    printf("WARNING :: nvcc -arch flag does not match actual device architecture\nWARNING :: The code can fail silently and produce wrong results\n");
    printf("Arch specified at compile = sm_%d Actual device = sm_%d\n",cudarch/10,actual/10);
  } 
#else
  printf("ERROR:: CUDA_ ARCH is not defined \n This should not be happening\n");
#endif
}
}
extern "C"{
void CudaCompileFlagCheck(){
  hypre_int devCount;
  cudaGetDeviceCount(&devCount);
  hypre_int i;
  hypre_int cudarch_actual;
  for(i = 0; i < devCount; ++i)
  {
    struct cudaDeviceProp props;
    cudaGetDeviceProperties(&props, i);
    cudarch_actual=props.major*100+props.minor*10;
  }
  hypre_CheckErrorDevice(cudaPeekAtLastError());
  hypre_CheckErrorDevice(cudaDeviceSynchronize());
  CompileFlagSafetyCheck<<<1,1,0,0>>>(cudarch_actual);
  cudaError_t code=cudaPeekAtLastError();
  if (code != cudaSuccess)
  {
    fprintf(stderr,"ERROR in CudaCompileFlagCheck%s \n", cudaGetErrorString(code));
    fprintf(stderr,"ERROR :: Check if compile arch flags match actual device arch = sm_%d\n",cudarch_actual/10);
    exit(2);
  }
  hypre_CheckErrorDevice(cudaDeviceSynchronize());
}
}

//written by KS
//naive version

extern "C"{
__global__
void MassInnerProdKernel(HYPRE_Real * __restrict__ u,  HYPRE_Real ** __restrict__ v, HYPRE_Real * result, HYPRE_Int k, HYPRE_Int n){
  hypre_int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i<n){
    // KS we should fetch u to shared or to registers 
    int j;	
#pragma unroll								
    for (j =0; j<k; ++j){
      //if ( blockIdx.x < 100000){printf("adding %f * %f  about to requrest v[%d][%d]\n", u[i], v[j][i], j, i);}
      //sum += u[i]*v[j][i];
      atomicAdd_system(&result[j], u[i]*v[j][i]);								
    }
  }
}
}

//v2
extern "C"{
__global__
void MassInnerProdKernel_v1(HYPRE_Real * __restrict__ u,  HYPRE_Real ** __restrict__ v, HYPRE_Real * result, HYPRE_Int k, HYPRE_Int n){
 hypre_int i = blockIdx.x * blockDim.x + threadIdx.x;
hypre_int tid = threadIdx.x;

  int j;	
HYPRE_Real r_uk[150];
volatile __shared__ HYPRE_Real s_u [BlockSize];
// __shared__ HYPRE_Real  s_v [BlockSize][MaxSpace];
s_u[tid] = u[i];
for (j=0; j<k; j++){
r_uk[j] = v[j][i];
}
__syncthreads();  
  

if (i<n){

#pragma unroll								
    for (j =0; j<k; ++j){
      atomicAdd_system(&result[j], u[i]*v[j][i]);								
    }
  }
}
}

//`MassInnerProd(int, int, double**, double*, double*)
extern "C"{
void MassInnerProd(HYPRE_Int n, HYPRE_Int k, HYPRE_Real **v, HYPRE_Real *u, HYPRE_Real * result){

  hypre_int num_threads=64;
  hypre_int num_blocks=n/num_threads+1;
  hypre_CheckErrorDevice(cudaDeviceSynchronize());
  printf("about to launch on (%d, %d), k = %d n = %d \n", num_blocks, num_threads, k, n);

  MassInnerProdKernel_v1<<<num_blocks, num_threads>>>(u, v, result, k,n);


  hypre_CheckErrorDevice(cudaDeviceSynchronize());

}
}
/**hypre_int num_rows,HYPRE_Complex alpha, HYPRE_Complex *a,hypre_int *ia, hypre_int *ja, HYPRE_Complex *x, HYPRE_Complex beta, HYPRE_Complex *y */
extern "C"{
__global__
void CSRMatvecTKernel_v1(HYPRE_Int num_rows, const HYPRE_Real * __restrict__ a, const HYPRE_Int * __restrict__ ia,const __restrict__  HYPRE_Int  * ja,const  HYPRE_Real * x, HYPRE_Real * y){

  /*
     i = 0; num_rows-1
     for (jj = A_i[i]; jj < A_i[i+1]; jj++)
     {
     j = A_j[jj];
     y_data[j] += A_data[jj] * x_data[i];
     }


   */

  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j;

  if (i<num_rows) {
    const double xx = x[i];
    for (j=ia[i]; j< ia[i+1]; j++){
      //    y[ja[j]] += a[j]*xx;

      if (abs(xx*a[j]) >  1e-16){
        //	atomicAdd(&y[ja[j]], 0.1f); 

        atomicAdd_system(&y[ja[j]], a[j]*xx);    

      }
    }
  }

}
}

//v2 shared memory for x


extern "C"{
__global__
void CSRMatvecTKernel_v2(HYPRE_Int num_rows, const HYPRE_Real * __restrict__ a, const HYPRE_Int * __restrict__ ia,const __restrict__  HYPRE_Int  * ja,const  HYPRE_Real * x, HYPRE_Real * y){


  int i = blockIdx.x*blockDim.x + threadIdx.x;
  int j=threadIdx.x;
  __shared__ HYPRE_Real s_x[64];


  if (i<num_rows) {
    s_x[j] = x[i];
    __syncthreads();
    const double xx = s_x[j];
    for (j=ia[i]; j< ia[i+1]; j++){

      if (abs(xx*a[j]) >  1e-16){

        atomicAdd_system(&y[ja[j]], a[j]*xx);    

      }
    }
  }

}
}
extern "C"{
void MatvecTCSR(hypre_int num_rows,HYPRE_Complex alpha, HYPRE_Complex *a,hypre_int *ia, hypre_int *ja, HYPRE_Complex *x, HYPRE_Complex beta, HYPRE_Complex *y){
  hypre_int num_threads=64;
  hypre_int num_blocks=num_rows/num_threads+1;
  //	printf("blocks: %d threads %d \n", num_blocks, num_threads);
#ifdef CATCH_LAUNCH_ERRORS
  hypre_CheckErrorDevice(cudaPeekAtLastError());
  hypre_CheckErrorDevice(cudaDeviceSynchronize());
#endif    

  CSRMatvecTKernel_v1<<<num_blocks,num_threads>>>(num_rows, a, ia, ja, x, y);
  cudaDeviceSynchronize();
#ifdef CATCH_LAUNCH_ERRORS
  hypre_CheckErrorDevice(cudaPeekAtLastError());
  hypre_CheckErrorDevice(cudaDeviceSynchronize());
#endif

}
}

extern "C"{
__global__
void ParRelaxKernel(
    HYPRE_Int n,
    HYPRE_Int relax_points,
    HYPRE_Int *__restrict__ cf_marker,
    HYPRE_Int *__restrict__ A_diag_i,
    HYPRE_Int *__restrict__ A_diag_j,
    HYPRE_Real *__restrict__ A_diag_data,
    HYPRE_Int *__restrict__ A_offd_i,
    HYPRE_Int *__restrict__ A_offd_j,
    HYPRE_Real *__restrict__ A_offd_data,
    HYPRE_Real *__restrict__ Vext_data,
    HYPRE_Real *__restrict__ f_data,
    HYPRE_Real *__restrict__ u_data){

  hypre_int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n &&
      cf_marker[i] == relax_points && 
      A_diag_data[A_diag_i[i]] != 0.0)
  {
    HYPRE_Real res = f_data[i];
    for (int jj = A_diag_i[i]+1; jj < A_diag_i[i+1]; jj++)
    {
      int ii = A_diag_j[jj];
      if (ii>=i){          
        res -= A_diag_data[jj] * u_data[ii];
      }}
    for (int jj = A_offd_i[i]; jj < A_offd_i[i+1]; jj++)
    {
      int ii = A_offd_j[jj];
      res -= A_offd_data[jj] * Vext_data[ii];
    }
    u_data[i] = res / A_diag_data[A_diag_i[i]];
  }
}

void ParRelax(
    HYPRE_Int n,
    HYPRE_Int relax_points,
    HYPRE_Int * __restrict__ cf_marker,
    HYPRE_Int * __restrict__ A_diag_i,
    HYPRE_Int * __restrict__ A_diag_j,
    HYPRE_Real * __restrict__ A_diag_data,
    HYPRE_Int * __restrict__ A_offd_i,
    HYPRE_Int * __restrict__ A_offd_j,
    HYPRE_Real * __restrict__ A_offd_data,
    HYPRE_Real * __restrict__ Vext_data,
    HYPRE_Real * __restrict__ f_data,
    HYPRE_Real * u_data) {

  hypre_int num_threads=128;
  hypre_int num_blocks=n / num_threads + 1;

  /*     HYPRE_Real * d_u_data_out = NULL;
         cudaMalloc(&d_u_data_out, n * sizeof(HYPRE_Real));

         cudaMemset(d_u_data_out, 0, n * sizeof(HYPRE_Real));
   */
  ParRelaxKernel<<<num_blocks, num_threads>>>(n, relax_points, cf_marker, A_diag_i, A_diag_j, A_diag_data, A_offd_i, A_offd_j, A_offd_data, Vext_data, f_data, u_data);

  /*
     cudaMemcpy(u_data, d_u_data_out, n * sizeof(HYPRE_Real), cudaMemcpyDeviceToDevice);

     cudaFree(d_u_data_out);
   */
}
}

//L1Jacobi

extern "C"{
__global__
void ParRelaxL1JacobiKernel(HYPRE_Int n,
    HYPRE_Real * l1_norms,	
    HYPRE_Real  relax_weight,
    HYPRE_Int * A_diag_i,
    HYPRE_Int * A_diag_j,
    HYPRE_Real * A_diag_data,
    HYPRE_Int * A_offd_i,
    HYPRE_Int * A_offd_j,
    HYPRE_Real *  A_offd_data,
    HYPRE_Real *  Vtemp_data,
    HYPRE_Real * Vext_data,
    HYPRE_Real * f_data,
    HYPRE_Real *  u_data){

  hypre_int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n){
    int ii, jj;
    HYPRE_Real res;
    if (A_diag_data[A_diag_i[i]] != 0.0)
    {
      res = f_data[i];
      for (jj = A_diag_i[i]; jj < A_diag_i[i+1]; jj++)
      {
        ii = A_diag_j[jj];
        res -= A_diag_data[jj] * Vtemp_data[ii];
      }
      for (jj = A_offd_i[i]; jj < A_offd_i[i+1]; jj++)
      {
        ii = A_offd_j[jj];
        res -= A_offd_data[jj] * Vext_data[ii];
      }
      u_data[i] += (relax_weight*res)/l1_norms[i];
    }


  }
}

/*
   void ParRelaxL1Jacobi(
   HYPRE_Int n,
   HYPRE_Real * __restrict__ l1_data,
   HYPRE_Real __restrict__ relax_weight,
   HYPRE_Int *__restrict__ A_diag_i,
   HYPRE_Int *__restrict__ A_diag_j,
   HYPRE_Real *__restrict__ A_diag_data,
   HYPRE_Int *__restrict__ A_offd_i,
   HYPRE_Int *__restrict__ A_offd_j,
   HYPRE_Real *__restrict__ A_offd_data,
   HYPRE_Real *__restrict__ Vtemp_data,
   HYPRE_Real *__restrict__ f_data,
   HYPRE_Real *__restrict__ u_data);
 */
/*(int, double*, double, int*, int*, double*, int*, int*, double*, double*, double*, double*)*/
void ParRelaxL1Jacobi(HYPRE_Int n,
    HYPRE_Real *  l1_data,	
    HYPRE_Real relax_weight,
    HYPRE_Int * A_diag_i,
    HYPRE_Int * A_diag_j,
    HYPRE_Real * A_diag_data,
    HYPRE_Int * A_offd_i,
    HYPRE_Int *  A_offd_j,
    HYPRE_Real *  A_offd_data,
    HYPRE_Real *  Vtemp_data,
    HYPRE_Real * Vext_data,
    HYPRE_Real * f_data,
    HYPRE_Real * u_data) {

  hypre_int num_threads=128;
  hypre_int num_blocks=n / num_threads + 1;

  /*     HYPRE_Real * d_u_data_out = NULL;
         cudaMalloc(&d_u_data_out, n * sizeof(HYPRE_Real));

         cudaMemset(d_u_data_out, 0, n * sizeof(HYPRE_Real));
   */
  ParRelaxL1JacobiKernel<<<num_blocks, num_threads>>>(n,l1_data,relax_weight, A_diag_i, A_diag_j, A_diag_data, A_offd_i, A_offd_j, A_offd_data, Vtemp_data, Vext_data, f_data, u_data);

  /*
     cudaMemcpy(u_data, d_u_data_out, n * sizeof(HYPRE_Real), cudaMemcpyDeviceToDevice);

     cudaFree(d_u_data_out);
   */
}


__global__
void ParRelaxL1JacobiCFKernel(HYPRE_Int n,
    HYPRE_Int * cf_marker,	
    HYPRE_Int  relax_points,
    HYPRE_Real relax_weight,
    HYPRE_Real * l1_norms,
    HYPRE_Int * A_diag_i,
    HYPRE_Int * A_diag_j,
    HYPRE_Real * A_diag_data,
    HYPRE_Int * A_offd_i,
    HYPRE_Int * A_offd_j,
    HYPRE_Real *  A_offd_data,
    HYPRE_Real *  Vtemp_data,
    HYPRE_Real * Vext_data,
    HYPRE_Real * f_data,
    HYPRE_Real *  u_data){

  hypre_int i = blockIdx.x * blockDim.x + threadIdx.x;

  /*-----------------------------------------------------------
   * If i is of the right type ( C or F ) and diagonal is
   * nonzero, relax point i; otherwise, skip it.
   *-----------------------------------------------------------*/
  HYPRE_Int ii,jj;
  HYPRE_Real res;
  if (i<n && cf_marker[i] == relax_points
      && A_diag_data[A_diag_i[i]] != 0.0f)
  {
    res = f_data[i];
    for (jj = A_diag_i[i]; jj < A_diag_i[i+1]; jj++)
    {
      ii = A_diag_j[jj];
      res -= A_diag_data[jj] * Vtemp_data[ii];
    }
    for (jj = A_offd_i[i]; jj < A_offd_i[i+1]; jj++)
    {
      ii = A_offd_j[jj];
      res -= A_offd_data[jj] * Vext_data[ii];
      //printf("i=%d, Vext_data[%d] =  %f \n", i, ii, Vext_data[ii]);
    }
    u_data[i] += (relax_weight * res)/l1_norms[i];
  }


}//kernel

void ParRelaxL1JacobiCF(HYPRE_Int n,
    HYPRE_Int *  cf_marker,	
    HYPRE_Int relax_points,
    HYPRE_Real  relax_weight,
    HYPRE_Real * l1_norms,
    HYPRE_Int * A_diag_i,
    HYPRE_Int * A_diag_j,
    HYPRE_Real * A_diag_data,
    HYPRE_Int * A_offd_i,
    HYPRE_Int *  A_offd_j,
    HYPRE_Real *  A_offd_data,
    HYPRE_Real *  Vtemp_data,
    HYPRE_Real * Vext_data,
    HYPRE_Real * f_data,
    HYPRE_Real * u_data) {

  hypre_int num_threads=128;
  hypre_int num_blocks=n / num_threads + 1;

  /*     HYPRE_Real * d_u_data_out = NULL;
         cudaMalloc(&d_u_data_out, n * sizeof(HYPRE_Real));

         cudaMemset(d_u_data_out, 0, n * sizeof(HYPRE_Real));
   */
  ParRelaxL1JacobiCFKernel<<<num_blocks, num_threads>>>(n,cf_marker,relax_points,relax_weight,l1_norms, A_diag_i, A_diag_j, A_diag_data, A_offd_i, A_offd_j, A_offd_data, Vtemp_data, Vext_data, f_data, u_data);

  /*
     cudaMemcpy(u_data, d_u_data_out, n * sizeof(HYPRE_Real), cudaMemcpyDeviceToDevice);

     cudaFree(d_u_data_out);
   */
}


}



//end of KS code

#endif
